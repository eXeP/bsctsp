#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <float.h>

#include "cuda_preprocessing.cuh"
#include "util.cuh"


__global__ void boruvka_smallest_kernel(int n, float* x, float* y, float* pi, int* component, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, int excluded_vertex) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = blockIdx.y * blockDim.x;
    __shared__ float shared_x[64];
    __shared__ float shared_y[64];
    __shared__ float shared_pi[64];
    __shared__ float shared_best[64];
    __shared__ float shared_best_j[64];
    __shared__ float shared_component[64];
    __shared__ float shared_component_i[64];
    if (j+threadIdx.x < n) {
        shared_x[threadIdx.x] = x[j+threadIdx.x];
        shared_y[threadIdx.x] = y[j+threadIdx.x];
        shared_component[threadIdx.x] = component[j+threadIdx.x];
        shared_pi[threadIdx.x] = pi[j+threadIdx.x];
    }
    if (i < n)
        shared_component_i[threadIdx.x] = component[i];
    shared_best[threadIdx.x] = FLT_MAX;
    shared_best_j[threadIdx.x] = 0;
    __syncthreads();
    if (i >= n)
        return;
    float best = FLT_MAX;
    int best_j = -1;
    int component_i = component[i];
    if (!(i == excluded_vertex || i >= n)) {
        float pi_i = pi[i], x_i = x[i], y_i = y[i];
        if (j + blockDim.x > n || (excluded_vertex >= j && excluded_vertex < j + blockDim.x) || (i >= j && i < j + blockDim.x)) {
            for (int k = 0; k < 64; ++k) {
                if (j+k == excluded_vertex || j+k >= n || shared_component[k] == component_i || j+k == i)
                    continue;
                float d_ij = pi_i + shared_pi[k];
                d_ij += (x_i - shared_x[k]) * (x_i - shared_x[k]);
                d_ij += (y_i - shared_y[k]) * (y_i - shared_y[k]);
                if (d_ij < best) {
                    best = d_ij;
                    best_j = j+k;
                }
            }
        } else {
            for (int k = 0; k < 64; ++k) {
                if (shared_component[k] == component_i)
                    continue;
                float d_ij = pi_i + shared_pi[k];
                d_ij += (x_i - shared_x[k]) * (x_i - shared_x[k]);
                d_ij += (y_i - shared_y[k]) * (y_i - shared_y[k]);
                if (d_ij < best) {
                    best = d_ij;
                    best_j = j+k;
                }
            }
        }
    }
    shared_best[threadIdx.x] = best;
    shared_best_j[threadIdx.x] = best_j;
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int k = 0; k < 64; ++k) {
            int tmp_i = blockIdx.x * blockDim.x + k;
            if (tmp_i >= n || tmp_i == excluded_vertex)
                continue;
            component_i = shared_component_i[k];
            best = shared_best[k];
            best_j = shared_best_j[k];
            if (best < component_best[component_i] || best == component_best[component_i]) {
                while (atomicExch(&component_lock[component_i], 1) != 0);
                if (best < component_best[component_i]) {
                    component_best[component_i] = best;
                    component_best_i[component_i] = tmp_i;
                    component_best_j[component_i] = best_j;
                } else if (abs(best - component_best[component_i]) < 0.0000001) {
                    int mi_c = min(tmp_i, best_j), ma_c = max(tmp_i, best_j);
                    int mi_o = min(component_best_i[component_i], component_best_j[component_i]), ma_o = max(component_best_i[component_i], component_best_j[component_i]);
                    if (mi_c < mi_o) {
                        component_best_i[component_i] = tmp_i;
                        component_best_j[component_i] = best_j;
                    } else if (mi_c == mi_o && ma_c < ma_o) {
                        component_best_i[component_i] = tmp_i;
                        component_best_j[component_i] = best_j;
                    }
                }
                component_lock[component_i] = 0;
                __threadfence();
            }
        }
    }
}

__global__ void boruvka_update_components(int n, 
    int* component, int* successor, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, 
    int* degrees, float* L_T, int* components, int excluded_vertex) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    int component_i = component[i];
    int component_j = component[component_best_j[component_i]];
    
    component[i] = successor[i];
    int vertex_ii = component_best_i[component_i];
    int vertex_ij = component_best_j[component_i];
    int vertex_jj = component_best_j[component_j];
    if (i == vertex_ii) {
        if (vertex_jj == i) {
            if (i < vertex_ij) {
                atomicSub(components, 1);
                atomicAdd(L_T, component_best[component_i]);
            }
        } else {
            atomicSub(components, 1);
            atomicAdd(L_T, component_best[component_i]);
            atomicAdd(&degrees[i], 1);
            atomicAdd(&degrees[vertex_ij], 1);
        }
        __threadfence();
    }
}

__global__ void boruvka_remove_cycles(int n, 
    int* component, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, 
    int* degrees, float* L_T, int* components, int excluded_vertex) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    int component_i = component[i];
    int component_j = component[component_best_j[component_i]];

    int vertex_ii = component_best_i[component_i];
    int vertex_ij = component_best_j[component_i];
    int vertex_jj = component_best_j[component_j];
    if (i == vertex_ii) {
        if (vertex_jj == i) {
            if (i < vertex_ij) {
                component_best_j[component_i] = i;
            }
        }
    }
}

__global__ void boruvka_pointer_doubling(int n, 
    int* component, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, 
    int* degrees, float* L_T, int* components, int excluded_vertex) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    component[i] = component[component_best_j[component[i]]];
    component[i] = component[component_best_j[component[i]]];
}

__global__ void excluded_vertex_add(int n, 
    float* x, float* y, float* pi, int excluded_vertex, float* closest, int* closest_i, int* lock) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    float d_ij = pi[excluded_vertex] + pi[i];
    d_ij += (x[excluded_vertex] - x[i]) * (x[excluded_vertex] - x[i]);
    d_ij += (y[excluded_vertex] - y[i]) * (y[excluded_vertex] - y[i]);
    if (d_ij < closest[0] || d_ij < closest[1]) {
        while (atomicExch(&lock[0], 1) != 0);
        if (d_ij < closest[0] && d_ij < closest[1]) {
            closest[1] = closest[0];
            closest_i[1] = closest_i[0];
            closest[0] = d_ij;
            closest_i[0] = i;
        } else if (d_ij < closest[1]) {
            closest[1] = d_ij;
            closest_i[1] = i;
        }
        lock[0] = 0;
        __threadfence();
    }
}

__global__ void excluded_vertex_set(float* closest, int* closest_i, int* degrees, float* L_T, int excluded_vertex) {
    degrees[excluded_vertex] += 2;
    degrees[closest_i[0]]++;
    degrees[closest_i[1]]++;
    L_T[0] += closest[0] + closest[1];
}


std::pair<float, std::vector<int>> gpu_boruvka_onetree(int n, float* Gx, float* Gy, float* Gpi) {
    int* Gcomponent = NULL;
    hipMalloc((void**)&Gcomponent, n * sizeof(int));
    std::vector<int> super_init;
    for (int i = 0; i < n; ++i)
        super_init.push_back(i);
    hipMemcpy(Gcomponent, super_init.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int* Gsuccessor = NULL;
    hipMalloc((void**)&Gsuccessor, n * sizeof(int));
    hipMemcpy(Gsuccessor, super_init.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int* Gvertex_lock = NULL;
    hipMalloc((void**)&Gvertex_lock, n * sizeof(int));
    hipMemset(Gvertex_lock, 0, n*sizeof(int));

    float* Gsmallest_add = NULL;
    hipMalloc((void**)&Gsmallest_add, n * sizeof(float));

    int* Gsmallest_i = NULL;
    hipMalloc((void**)&Gsmallest_i, n * sizeof(int));
    int* Gsmallest_j = NULL;
    hipMalloc((void**)&Gsmallest_j, n * sizeof(int));

    int* Gdegrees = NULL;
    hipMalloc((void**)&Gdegrees, n * sizeof(int));
    hipMemset(Gdegrees, 0, n*sizeof(int));

    float* GL_T = NULL;
    hipMalloc((void**)&GL_T, 1 * sizeof(float));
    hipMemset(GL_T, 0, 1*sizeof(float));

    int* Gcomponents = NULL;
    hipMalloc((void**)&Gcomponents, 1 * sizeof(int));
    int tmp = n-1;
    hipMemcpy(Gcomponents, &tmp, 1 * sizeof(int), hipMemcpyHostToDevice);

    int excluded_vertex = 0;
    int components = n-1;
    dim3 dimBlock(64, 1);
    dim3 dimGrid(divup(n, 64), divup(n, 64));

    std::vector<float> inf(n, std::numeric_limits<float>::max());
    while (components > 1) {
        hipMemcpy(Gsmallest_add, inf.data(), n * sizeof(float), hipMemcpyHostToDevice);
        boruvka_smallest_kernel<<<dimGrid, dimBlock>>>(n, Gx, Gy, Gpi, Gcomponent, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, excluded_vertex);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        boruvka_remove_cycles<<<dim3(n, 1), dim3(1, 1)>>>(n, Gcomponent, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, Gdegrees, GL_T, Gcomponents, excluded_vertex);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        int n_pd = components;
        while (n_pd > 0) {
            boruvka_pointer_doubling<<<dim3(n, 1), dim3(1, 1)>>>(n, Gsuccessor, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, Gdegrees, GL_T, Gcomponents, excluded_vertex);
            CHECK(hipGetLastError());
            hipDeviceSynchronize();
            n_pd /= 2;
        }
        boruvka_update_components<<<dim3(n, 1), dim3(1, 1)>>>(n, Gcomponent, Gsuccessor, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, Gdegrees, GL_T, Gcomponents, excluded_vertex);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        hipMemcpy(&components, Gcomponents, 1 * sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
    }
    float* Gclosest = NULL;
    hipMalloc((void**)&Gclosest, 2 * sizeof(float));
    hipMemcpy(Gclosest, inf.data(), 2 * sizeof(float), hipMemcpyHostToDevice);
    int* Gclosest_idx = NULL;
    hipMalloc((void**)&Gclosest_idx, 2 * sizeof(int));
    excluded_vertex_add<<<dim3(n, 1), dim3(1, 1)>>>(n, Gx, Gy, Gpi, excluded_vertex, Gclosest, Gclosest_idx, Gvertex_lock);
    CHECK(hipGetLastError());
    hipDeviceSynchronize();
    excluded_vertex_set<<<dim3(1, 1), dim3(1, 1)>>>(Gclosest, Gclosest_idx, Gdegrees, GL_T, excluded_vertex);
    CHECK(hipGetLastError());
    hipDeviceSynchronize();

    float return_length;
    hipMemcpy(&return_length, GL_T, 1 * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::vector<int> d(n);
    hipMemcpy(d.data(), Gdegrees, n * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(Gcomponent);
    hipFree(Gsuccessor);
    hipFree(Gvertex_lock);
    hipFree(Gsmallest_add);
    hipFree(Gsmallest_i);
    hipFree(Gsmallest_j);
    hipFree(Gdegrees);
    hipFree(GL_T);
    hipFree(Gcomponents);
    hipFree(Gclosest);
    hipFree(Gclosest_idx);
    return {return_length, d};
}

std::vector<float> gpu_subgradient_opt_alpha(float* x, float* y, int n) {
    printf("\n----Subgradient opt. for alpha-nearness on GPU----\n");
    float* Gpi = NULL;
    hipMalloc((void**)&Gpi, n * sizeof(float));
    hipMemset(Gpi, 0, n*sizeof(float));
    float* Gx = NULL;
    hipMalloc((void**)&Gx, n * sizeof(float));
    hipMemcpy(Gx, x, n * sizeof(float), hipMemcpyHostToDevice);
    float* Gy = NULL;
    hipMalloc((void**)&Gy, n * sizeof(float));
    hipMemcpy(Gy, y, n * sizeof(float), hipMemcpyHostToDevice);

    std::vector<float> pi(n, 0), best_pi(n, 0);
    auto [init_w, init_d] = gpu_boruvka_onetree(n, Gx, Gy, Gpi);
    float best_w = init_w;
    std::vector<int> last_v(n), v(n);
    bool is_tour = true;
    for (int i = 0; i < n; ++i) {
        last_v[i] = init_d[i] - 2;
        v[i] = last_v[i];
        is_tour &= (last_v[i] == 0);
    }
    bool initial_phase = true;
    int initial_period = max(n/2, 100);
    int period = initial_period;
    for (float t = 1.f; t > 0; t /= 2.f, period /= 2) {
        for (int p = 1; t > 0 && p <= period; ++p) {
            for (int i = 0; i < n; ++i) {
                pi[i] += t * ( 0.7f * v[i] + 0.3f * last_v[i]);
            }
            hipMemcpy(Gpi, pi.data(), n * sizeof(float), hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            last_v = v;
            auto [w, d] = gpu_boruvka_onetree(n, Gx, Gy, Gpi);
            is_tour = true;
            for (int i = 0; i < n; ++i) {
                v[i] = d[i] - 2;
                is_tour &= (v[i] == 0);
            }
            if (w > best_w) {
                best_w = w;
                best_pi = pi;
                if (initial_phase)
                    t *= 2.f;
                if (p == period)
                    period *= 2;
            } else if (initial_phase && p > initial_period / 2) {
                initial_phase = false;
                p = 0;
                t = 0.75f * t;
            }
        }
    }
    hipFree(Gpi);
    hipFree(Gx);
    hipFree(Gy);
    return best_pi;
}
