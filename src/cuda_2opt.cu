#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <float.h>

#include "cuda_2opt.cuh"
#include "util.cuh"


struct best_struct {
    float best;
    int i;
    int j;
};

__global__ void two_opt_kernel(const float* x, const float* y, best_struct* return_best, int n, int* lock) {
    int i = threadIdx.x + blockIdx.x * blockDim.x+1;
    int j = blockIdx.y * blockDim.x;
    __shared__ float shared_x[65];
    __shared__ float shared_y[65];
    __shared__ float shared_best[64];
    __shared__ int shared_j[64];
    shared_best[threadIdx.x] = 0;
    shared_j[threadIdx.x] = 0;
    if (j+threadIdx.x < n) {
        shared_x[threadIdx.x] = x[j+threadIdx.x];
        shared_y[threadIdx.x] = y[j+threadIdx.x];
    }
    if (threadIdx.x == blockDim.x-1) {
        shared_x[blockDim.x] = x[j+threadIdx.x+1];
        shared_y[blockDim.x] = y[j+threadIdx.x+1];
    }
    if (j+blockDim.x+1 <= i || i >= n-2)
        return;
    __syncthreads();
    float best = 0;
    int best_j = 0;
    if (i > 0) {
        float xi = x[i], xim = x[i-1], yi = y[i], yim = y[i-1];
        float i_dist = dist(xi, yi, xim, yim);
        if (j <= i || j + blockDim.x >= n) {
            for (int k = 0; k < blockDim.x; ++k) {
                if (j+k > i && j+k < n-1) {
                    float k_dist = i_dist + dist(shared_x[k], shared_y[k], shared_x[k+1], shared_y[k+1]) - 
                    (dist(xi, yi, shared_x[k+1], shared_y[k+1]) + dist(xim, yim, shared_x[k], shared_y[k]));
                    if (k_dist > best) {
                        best = k_dist;
                        best_j = j+k;
                    }
                }
            }
        } else {
            for (int k = 0; k < blockDim.x; ++k) {
                float k_dist = i_dist + dist(shared_x[k], shared_y[k], shared_x[k+1], shared_y[k+1]) - 
                (dist(xi, yi, shared_x[k+1], shared_y[k+1]) + dist(xim, yim, shared_x[k], shared_y[k]));
                if (k_dist > best) {
                    best = k_dist;
                    best_j = j+k;
                }
            }
        }
    }

    int out_w = divupg(n, 64);
    shared_best[threadIdx.x] = best;
    shared_j[threadIdx.x] = best_j;
    int best_i = i;
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int i2 = 0; i2 < blockDim.x; ++i2) {
            if (i + i2 >= n-2)
                continue;
            if (shared_best[i2] > best) {
                best_i = i+i2;
                best = shared_best[i2];
                best_j = shared_j[i2];
            }
        }
        if (best > return_best[0].best) {
            while (atomicExch(&lock[0], 1) != 0);
            if (best > return_best[0].best) {
                return_best[0].best = best;
                return_best[0].i = best_i;
                return_best[0].j = best_j;
            }
            lock[0] = 0;
            __threadfence();
        }
    }
}

__global__ void two_opt_swap_kernel(float* x, float* y, int i, int j) {
    int k = blockIdx.x;
    float tmp = x[i+k];
    x[i+k] = x[j-k];
    x[j-k] = tmp;
    tmp = y[i+k];
    y[i+k] = y[j-k];
    y[j-k] = tmp;
}

void run_gpu_2opt(float* x, float* y, int n) {
    float* xGPU = NULL;
    hipMalloc((void**)&xGPU, n * sizeof(float));
    hipMemcpy(xGPU, x, n * sizeof(float), hipMemcpyHostToDevice);
    float* yGPU = NULL;
    hipMalloc((void**)&yGPU, n * sizeof(float));
    hipMemcpy(yGPU, y, n * sizeof(float), hipMemcpyHostToDevice);
    best_struct* bestGPU = NULL;
    hipMalloc((void**)&bestGPU, sizeof(best_struct));
    int* lock = NULL;
    hipMalloc((void**)&lock, 1 * sizeof(int));

    dim3 dimBlock(64, 1);
    dim3 dimGrid(divup(n, 64), divup(n, 64));
    do {
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        hipMemset(bestGPU, 0, sizeof(best_struct));
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        hipMemset(lock, 0, 1*sizeof(int));
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        two_opt_kernel<<<dimGrid, dimBlock>>>(xGPU, yGPU, bestGPU, n, lock);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        best_struct* best = (best_struct*)malloc(sizeof(best_struct));
        hipMemcpy(best, bestGPU, sizeof(best_struct), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        if (abs(best[0].best) < 0.000001)
            break;
        //printf("Improvement %f %d %d\n", best[0].best, best[0].i, best[0].j);
        two_opt_swap_kernel<<<dim3((best[0].j-best[0].i+1)/2, 1), dim3(1, 1)>>>(xGPU, yGPU, best[0].i, best[0].j);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
    } while(true);

    hipMemcpy(x, xGPU, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(y, yGPU, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(xGPU);
    hipFree(yGPU);
    hipFree(bestGPU);
    hipFree(lock);
}