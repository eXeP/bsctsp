#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <float.h>

#include "tsp_preprocessing.cuh"
#include "util.cuh"


__global__ void boruvka_smallest_kernel(int n, float* x, float* y, float* pi, int* component, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, int excluded_vertex) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = blockIdx.y * blockDim.x;
    __shared__ float shared_x[64];
    __shared__ float shared_y[64];
    __shared__ float shared_pi[64];
    __shared__ float shared_best[64];
    __shared__ float shared_best_j[64];
    __shared__ float shared_component[64];
    __shared__ float shared_component_i[64];
    if (j+threadIdx.x < n) {
        shared_x[threadIdx.x] = x[j+threadIdx.x];
        shared_y[threadIdx.x] = y[j+threadIdx.x];
        shared_component[threadIdx.x] = component[j+threadIdx.x];
        shared_pi[threadIdx.x] = pi[j+threadIdx.x];
    }
    if (i < n)
        shared_component_i[threadIdx.x] = component[i];
    shared_best[threadIdx.x] = FLT_MAX;
    shared_best_j[threadIdx.x] = 0;
    __syncthreads();
    if (i >= n)
        return;
    float best = FLT_MAX;
    int best_j = -1;
    int component_i = component[i];
    if (!(i == excluded_vertex || i >= n)) {
        float pi_i = pi[i], x_i = x[i], y_i = y[i];
        if (j + blockDim.x > n || (excluded_vertex >= j && excluded_vertex < j + blockDim.x) || (i >= j && i < j + blockDim.x)) {
            for (int k = 0; k < 64; ++k) {
                if (j+k == excluded_vertex || j+k >= n || shared_component[k] == component_i || j+k == i)
                    continue;
                float d_ij = pi_i + shared_pi[k];
                d_ij += (x_i - shared_x[k]) * (x_i - shared_x[k]) + (y_i - shared_y[k]) * (y_i - shared_y[k]);
                if (d_ij < best) {
                    best = d_ij;
                    best_j = j+k;
                }
            }
        } else {
            for (int k = 0; k < 64; ++k) {
                if (shared_component[k] == component_i)
                    continue;
                float d_ij = pi_i + shared_pi[k];
                d_ij += (x_i - shared_x[k]) * (x_i - shared_x[k]) + (y_i - shared_y[k]) * (y_i - shared_y[k]);
                if (d_ij < best) {
                    best = d_ij;
                    best_j = j+k;
                }
            }
        }
    }
    shared_best[threadIdx.x] = best;
    shared_best_j[threadIdx.x] = best_j;
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int k = 0; k < 64; ++k) {
            int tmp_i = blockIdx.x * blockDim.x + k;
            if (tmp_i >= n || tmp_i == excluded_vertex)
                continue;
            component_i = shared_component_i[k];
            best = shared_best[k];
            best_j = shared_best_j[k];
            if (best < component_best[component_i] || best == component_best[component_i]) {
                while (atomicExch(&component_lock[component_i], 1) != 0);
                if (best < component_best[component_i]) {
                    component_best[component_i] = best;
                    component_best_i[component_i] = tmp_i;
                    component_best_j[component_i] = best_j;
                } else if (abs(best - component_best[component_i]) < 0.0000001) {
                    int mi_c = min(tmp_i, best_j), ma_c = max(tmp_i, best_j);
                    int mi_o = min(component_best_i[component_i], component_best_j[component_i]), ma_o = max(component_best_i[component_i], component_best_j[component_i]);
                    if (mi_c < mi_o) {
                        component_best_i[component_i] = tmp_i;
                        component_best_j[component_i] = best_j;
                    } else if (mi_c == mi_o && ma_c < ma_o) {
                        component_best_i[component_i] = tmp_i;
                        component_best_j[component_i] = best_j;
                    }
                }
                component_lock[component_i] = 0;
                __threadfence();
            }
        }
    }
}

__global__ void boruvka_update_components(int n, 
    int* component, int* successor, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, 
    int* degrees, float* L_T, int* components, int excluded_vertex) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    int component_i = component[i];
    int component_j = component[component_best_j[component_i]];
    
    int component_min = min(component_i, component_j);
    component[i] = successor[i];
    int vertex_ii = component_best_i[component_i];
    int vertex_ji = component_best_i[component_j];
    int vertex_ij = component_best_j[component_i];
    int vertex_jj = component_best_j[component_j];
    if (i == vertex_ii) {
        if (vertex_jj == i) {
            if (i < vertex_ij) {
                atomicSub(components, 1);
                atomicAdd(L_T, component_best[component_i]);
            }
        } else {
            atomicSub(components, 1);
            atomicAdd(L_T, component_best[component_i]);
            atomicAdd(&degrees[i], 1);
            atomicAdd(&degrees[vertex_ij], 1);
        }
        __threadfence();
    }
}

__global__ void boruvka_remove_cycles(int n, 
    int* component, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, 
    int* degrees, float* L_T, int* components, int excluded_vertex) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    int component_i = component[i];
    int component_j = component[component_best_j[component_i]];

    int vertex_ii = component_best_i[component_i];
    int vertex_ji = component_best_i[component_j];
    int vertex_ij = component_best_j[component_i];
    int vertex_jj = component_best_j[component_j];
    if (i == vertex_ii) {
        if (vertex_jj == i) {
            if (i < vertex_ij) {
                component_best_j[component_i] = i;
            }
        }
    }
}

__global__ void boruvka_pointer_doubling(int n, 
    int* component, float* component_best, int* component_best_i, int* component_best_j, int* component_lock, 
    int* degrees, float* L_T, int* components, int excluded_vertex) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    component[i] = component[component_best_j[component[i]]];
    component[i] = component[component_best_j[component[i]]];
}

__global__ void excluded_vertex_add(int n, 
    float* x, float* y, float* pi, int excluded_vertex, float* closest, int* closest_i, int* lock) {
    int i = blockIdx.x;
    if (i == excluded_vertex)
        return;
    float d_ij = pi[excluded_vertex] + pi[i];
    d_ij += (x[excluded_vertex] - x[i]) * (x[excluded_vertex] - x[i]);
    d_ij += (y[excluded_vertex] - y[i]) * (y[excluded_vertex] - y[i]);
    if (d_ij < closest[0] || d_ij < closest[1]) {
        while (atomicExch(&lock[0], 1) != 0);
        if (d_ij < closest[0] && d_ij < closest[1]) {
            closest[1] = closest[0];
            closest_i[1] = closest_i[0];
            closest[0] = d_ij;
            closest_i[0] = i;
        } else if (d_ij < closest[1]) {
            closest[1] = d_ij;
            closest_i[1] = i;
        }
        lock[0] = 0;
    }
}

__global__ void excluded_vertex_set(float* closest, int* closest_i, int* degrees, float* L_T, int excluded_vertex) {
    degrees[excluded_vertex] += 2;
    degrees[closest_i[0]]++;
    degrees[closest_i[1]]++;
    L_T[0] += closest[0] + closest[1];
}


std::pair<float, std::vector<int>> gpu_boruvka_onetree(int n, float* Gx, float* Gy, float* Gpi) {
    int* Gcomponent = NULL;
    hipMalloc((void**)&Gcomponent, n * sizeof(int));
    std::vector<int> super_init;
    for (int i = 0; i < n; ++i)
        super_init.push_back(i);
    hipMemcpy(Gcomponent, super_init.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int* Gsuccessor = NULL;
    hipMalloc((void**)&Gsuccessor, n * sizeof(int));
    hipMemcpy(Gsuccessor, super_init.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int* Gvertex_lock = NULL;
    hipMalloc((void**)&Gvertex_lock, n * sizeof(int));
    hipMemset(Gvertex_lock, 0, n*sizeof(int));

    float* Gsmallest_add = NULL;
    hipMalloc((void**)&Gsmallest_add, n * sizeof(float));


    int* Gsmallest_i = NULL;
    hipMalloc((void**)&Gsmallest_i, n * sizeof(int));
    int* Gsmallest_j = NULL;
    hipMalloc((void**)&Gsmallest_j, n * sizeof(int));

    int* Gdegrees = NULL;
    hipMalloc((void**)&Gdegrees, n * sizeof(int));
    hipMemset(Gdegrees, 0, n*sizeof(int));

    float* GL_T = NULL;
    hipMalloc((void**)&GL_T, 1 * sizeof(float));
    hipMemset(GL_T, 0, 1*sizeof(float));

    int* Gcomponents = NULL;
    hipMalloc((void**)&Gcomponents, 1 * sizeof(int));
    int tmp = n-1;
    hipMemcpy(Gcomponents, &tmp, 1 * sizeof(int), hipMemcpyHostToDevice);

    int excluded_vertex = 0;
    int components = n-1;
    dim3 dimBlock(64, 1);
    dim3 dimGrid(divup(n, 64), divup(n, 64));

    std::vector<float> inf(n, std::numeric_limits<float>::max());
    while (components > 1) {
        hipMemset(Gsmallest_i, 0, n*sizeof(int));
        hipMemset(Gsmallest_j, 0, n*sizeof(int));
        hipMemcpy(Gsmallest_add, inf.data(), n * sizeof(float), hipMemcpyHostToDevice);
        boruvka_smallest_kernel<<<dimGrid, dimBlock>>>(n, Gx, Gy, Gpi, Gcomponent, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, excluded_vertex);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        boruvka_remove_cycles<<<dim3(n, 1), dim3(1, 1)>>>(n, Gcomponent, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, Gdegrees, GL_T, Gcomponents, excluded_vertex);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        int n_pd = components;
        while (n_pd > 0) {
            boruvka_pointer_doubling<<<dim3(n, 1), dim3(1, 1)>>>(n, Gsuccessor, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, Gdegrees, GL_T, Gcomponents, excluded_vertex);
            CHECK(hipGetLastError());
            hipDeviceSynchronize();
            n_pd /= 2;
        }
        boruvka_update_components<<<dim3(n, 1), dim3(1, 1)>>>(n, Gcomponent, Gsuccessor, Gsmallest_add, Gsmallest_i, Gsmallest_j, Gvertex_lock, Gdegrees, GL_T, Gcomponents, excluded_vertex);
        CHECK(hipGetLastError());
        hipDeviceSynchronize();
        hipMemcpy(&components, Gcomponents, 1 * sizeof(int), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        //int tmp;
        //std::cin >> tmp;
    }
    float* Gclosest = NULL;
    hipMalloc((void**)&Gclosest, 2 * sizeof(float));
    hipMemcpy(Gclosest, inf.data(), 2 * sizeof(float), hipMemcpyHostToDevice);
    int* Gclosest_idx = NULL;
    hipMalloc((void**)&Gclosest_idx, 2 * sizeof(int));
    excluded_vertex_add<<<dim3(n, 1), dim3(1, 1)>>>(n, Gx, Gy, Gpi, excluded_vertex, Gclosest, Gclosest_idx, Gvertex_lock);
    CHECK(hipGetLastError());
    hipDeviceSynchronize();
    excluded_vertex_set<<<dim3(1, 1), dim3(1, 1)>>>(Gclosest, Gclosest_idx, Gdegrees, GL_T, excluded_vertex);
    CHECK(hipGetLastError());
    hipDeviceSynchronize();

    float return_length;
    hipMemcpy(&return_length, GL_T, 1 * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::vector<int> d(n);
    hipMemcpy(d.data(), Gdegrees, n * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    return {return_length, d};
}

std::vector<float> gpu_subgradient_opt_alpha(float* x, float* y, int n) {
    printf("\n----GPU----\n");
    std::vector<float> pi(n, 0);
    float* Gpi = NULL;
    hipMalloc((void**)&Gpi, n * sizeof(float));
    hipMemset(Gpi, 0, n*sizeof(float));
    float* Gx = NULL;
    hipMalloc((void**)&Gx, n * sizeof(float));
    hipMemcpy(Gx, x, n * sizeof(float), hipMemcpyHostToDevice);
    float* Gy = NULL;
    hipMalloc((void**)&Gy, n * sizeof(float));
    hipMemcpy(Gy, y, n * sizeof(float), hipMemcpyHostToDevice);

    float W = -1<<28;
    float t = 1.0;
    int period = n/2;
    int np = 4;
    while (true) {
        const auto& [length, d] = gpu_boruvka_onetree(n, Gx, Gy, Gpi);
        float w = length;
        for (int i = 0; i < n; ++i)
            w -= pi[i];
        W = std::max(W, w);
        bool is_tour = true;
        std::vector<int> v(n);
        for (int i = 0; i < n; ++i) {
            v[i] = d[i] - 2;
            is_tour &= v[i] == 0;
        }
        for (int i = 0; i < n; ++i) {
            pi[i] = pi[i] + t * v[i];
        }
        period--;
        if (period == 0) {
            t *= 0.5;
            period = n/np;
            np *= 2;
        }
        std::cout << is_tour << " " << t << " " << period << " " << length << std::endl;
        if (is_tour || t < 0.001 || period == 0) 
            break;
        hipMemcpy(Gpi, pi.data(), n * sizeof(float), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
    }
    std::cout << "Done, pi:" << std::endl;
    for (int i = 0; i < n; ++i)
        std::cout << pi[i] << " ";
    std::cout << std::endl;
    return pi;
}
